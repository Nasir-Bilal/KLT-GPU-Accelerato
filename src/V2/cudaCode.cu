#include "hip/hip_runtime.h"
// src/V2/cudaCode.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#include "cudaCode.h"

// --- GPU kernel (runs on device) ---
__global__ void exampleKernel(float *d_data, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = d_data[idx];
        // simple demo computation
        d_data[idx] = x * x + 1.0f;
    }
}

// --- Host wrapper callable from C ---
extern "C" void runCudaExampleKernel(float *data, int n)
{
    float *d_data;
    size_t bytes = sizeof(float) * n;

    // Allocate device memory
    hipMalloc(&d_data, bytes);

    // Copy data to device
    hipMemcpy(d_data, data, bytes, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize  = (n + blockSize - 1) / blockSize;

    exampleKernel<<<gridSize, blockSize>>>(d_data, n);
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(data, d_data, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);

    printf("[CUDA] Kernel executed successfully on %d elements\n", n);
}
    
/*--------------- _convolveImageHoriz ---------------*/

#define CUDA_CHECK(call) \
  do { hipError_t e = (call); if (e != hipSuccess) { \
    fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); exit(1); } } while(0)

typedef struct {
    float *data;  
    int ncols;
    int nrows;
} KLT_FloatImage;

typedef struct {
    float *data;   
    int width;
} ConvolutionKernel;

//KERNEL
__global__ void convolve_horiz_kernel(const float* imgin,
                           const float* kernel_data,
                           float* imgout,
                           int ncols, int nrows,
                           int kernelWidth)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= nrows || col >= ncols) return;

    int radius = kernelWidth / 2;

    // Replicate CPU border handling: zero left/right columns within radius
    if (col < radius || col >= (ncols - radius)) {
        imgout[row * ncols + col] = 0.0f;
        return;
    }

    // ppp = ptrrow + i - radius;
    // sum = 0.0;
    // for (k = kernel.width-1 ; k >= 0 ; k--)
    //   sum += *ppp++ * kernel.data[k];
    float sum = 0.0f;
    int p = col - radius;                    // p corresponds to offset (i - radius)
    int rowBase = row * ncols;
    // iterate k from kernelWidth-1 down to 0 to match original ordering exactly
    for (int k = kernelWidth - 1; k >= 0; --k) {
        float v = imgin[rowBase + p];       // *ppp
        float w = kernel_data[k];           // kernel.data[k]
        sum += v * w;
        ++p;                                // ppp++
    }

    imgout[rowBase + col] = sum;
}

//WRAPPER
void convolve_horiz_cuda(const float* h_imgin, const float* h_kernel, float* h_imgout, int ncols, int nrows, int kernelWidth)
{
    assert(kernelWidth % 2 == 1);                  // same assertion as CPU
    // We'll assume imgin != imgout on the host; this wrapper requires separate buffers.

    size_t imgBytes = (size_t)ncols * nrows * sizeof(float);
    size_t kernelBytes = (size_t)kernelWidth * sizeof(float);

    float *d_imgin = nullptr, *d_imgout = nullptr, *d_kernel = nullptr;
    CUDA_CHECK(hipMalloc(&d_imgin, imgBytes));
    CUDA_CHECK(hipMalloc(&d_imgout, imgBytes));
    CUDA_CHECK(hipMalloc(&d_kernel, kernelBytes));

    CUDA_CHECK(hipMemcpy(d_imgin, h_imgin, imgBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, kernelBytes, hipMemcpyHostToDevice));
    // Note: we don't initialize d_imgout (kernel writes every pixel)

    // Choose a simple 2D block size
    dim3 block(16, 16); // simple choice, easy to understand
    dim3 grid( (ncols + block.x - 1) / block.x,
               (nrows + block.y - 1) / block.y );

    convolve_horiz_kernel<<<grid, block>>>(d_imgin, d_kernel, d_imgout, ncols, nrows, kernelWidth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_imgout, d_imgout, imgBytes, hipMemcpyDeviceToHost));

    hipFree(d_imgin);
    hipFree(d_imgout);
    hipFree(d_kernel);
}

/*--------------- _convolveImageVert ---------------*/

//KERNEL
__global__ void convolve_vert_kernel(const float* __restrict__ imgin, const float* __restrict__ kernel_data, float* __restrict__ imgout, int ncols, int nrows, int kernelWidth)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= ncols || row >= nrows) return;

    int radius = kernelWidth / 2;

    // Border handling: top/bottom rows within radius are zero
    if (row < radius || row >= (nrows - radius)) {
        imgout[row * ncols + col] = 0.0f;
        return;
    }

    // Compute convolution in column direction.
    // Mirror CPU order: for (k = kernel.width-1; k >= 0; k--) sum += *ppp++ * kernel.data[k];
    float sum = 0.0f;
    int p_row = row - radius;        // starting row index for ppp
    int base_index = col;            // column offset for indexing: idx = p_row * ncols + col
    for (int k = kernelWidth - 1; k >= 0; --k) {
        float v = imgin[p_row * ncols + base_index]; // *ppp
        float w = kernel_data[k];
        sum += v * w;
        ++p_row; // ppp += ncols (move one row down)
    }

    imgout[row * ncols + col] = sum;
}

//WRAPPER
void convolve_vert_cuda(const float* h_imgin, const float* h_kernel, float* h_imgout, int ncols, int nrows, int kernelWidth)
{
    assert(kernelWidth % 2 == 1);

    size_t imgBytes = (size_t)ncols * (size_t)nrows * sizeof(float);
    size_t kernelBytes = (size_t)kernelWidth * sizeof(float);

    float *d_imgin = NULL, *d_imgout = NULL, *d_kernel = NULL;
    CUDA_CHECK(hipMalloc(&d_imgin, imgBytes));
    CUDA_CHECK(hipMalloc(&d_imgout, imgBytes));
    CUDA_CHECK(hipMalloc(&d_kernel, kernelBytes));

    CUDA_CHECK(hipMemcpy(d_imgin, h_imgin, imgBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, kernelBytes, hipMemcpyHostToDevice));

    // choose simple 2D block/grid
    dim3 block(16, 16);
    dim3 grid( (ncols + block.x - 1) / block.x,
               (nrows + block.y - 1) / block.y );

    convolve_vert_kernel<<<grid, block>>>(d_imgin, d_kernel, d_imgout, ncols, nrows, kernelWidth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_imgout, d_imgout, imgBytes, hipMemcpyDeviceToHost));

    hipFree(d_imgin);
    hipFree(d_imgout);
    hipFree(d_kernel);
}