#include "hip/hip_runtime.h"
// src/V2/cudaCode.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include "cudaCode.h"

// --- GPU kernel (runs on device) ---
__global__ void exampleKernel(float *d_data, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = d_data[idx];
        // simple demo computation
        d_data[idx] = x * x + 1.0f;
    }
}

// --- Host wrapper callable from C ---
extern "C" void runCudaExampleKernel(float *data, int n)
{
    float *d_data;
    size_t bytes = sizeof(float) * n;

    // Allocate device memory
    hipMalloc(&d_data, bytes);

    // Copy data to device
    hipMemcpy(d_data, data, bytes, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize  = (n + blockSize - 1) / blockSize;

    exampleKernel<<<gridSize, blockSize>>>(d_data, n);
    hipDeviceSynchronize();

    // Copy results back
    hipMemcpy(data, d_data, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);

    printf("[CUDA] Kernel executed successfully on %d elements\n", n);
}
    